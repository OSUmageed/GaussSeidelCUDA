#include "hip/hip_runtime.h"
//THE COPY SHOULD WORK IN LINUX!

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"
#include "hip/hip_runtime_api.h"
#include "hip/driver_types.h"

#include <thrust/reduce.h>
#include <thrust/execution_policy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/for_each.h>
#include <thrust/transform.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/copy.h>

#include <stdio.h>
#include <cmath>
#include <time.h>
#include <stdlib.h>
#include <fstream>
#include <iostream>
#include <typeinfo>

using namespace std;

// Now the plate must be square.  #Divisions should be some multiple of 32, preferably some 2^x.
#define LENS       5.
#define TH_COND   16.
#define DZ        .01
#define DIVISIONS 1024.
#define TOLERANCE 1.e-2
#define REAL float

struct absdiff
{
    template <typename Tuple>
    __host__ __device__
    void operator()(Tuple t)
    {
        // diffmat = (redi-redf)+blacki-blackf)
        thrust::get<4>(t) = fabsf(thrust::get<0>(t) - thrust::get<1>(t)) + fabsf(thrust::get<2>(t) - thrust::get<3>(t));
    }
};

__device__ void cornerSource (REAL BC1, REAL BC2, REAL *source, REAL coff)
{

    if (BC1>0)
    {
        if (BC2>0)
        {
            source[0] = 2.0f * coff * (BC1 + BC2);
            source[1] = 4.0f * coff;
        }
        else
        {
            source[0] = 2.0f * coff * BC1;
            source[1] = 2.0f * coff;
        }
    }

    else if (BC2>0)
    {
        source[0] = 2.0f * coff * BC2;
        source[1] = 2.0f * coff;
    }
    else
    {
        source[0] = 0.0f;
        source[1] = 0.0f;

    }

}

__global__ void differencingOperation(REAL *active_half, REAL *passive_half, REAL *d_const, const int turn)
{
	int ind_x = blockIdx.x * blockDim.x + threadIdx.x;
	int ind_y = blockIdx.y * blockDim.y + threadIdx.y;
	int id = ind_x + ind_y * int(DIVISIONS*.5);
	int grd;
	grd = int(DIVISIONS*.5);
	__shared__ int seq;

    REAL d_coeff_p;
    REAL *source = new REAL[2];

	// Negative seq means active half starts first.  Positive seq means passive half starts first.
	if (((turn + ind_y) & 1) == 0)
	{
		seq = -1;
	}
	else
	{
		seq = 1;
	}


	// If bottom row.
	if (id < DIVISIONS*DIVISIONS*.5)
	{
	if (ind_y == 0)
	{
		// If bottom left (SouthWest) corner and red.
		if (ind_x == 0 && turn == 0)
		{
			cornerSource(d_const[2],d_const[3], source, d_const[4]);
			d_coeff_p = 2.0f * d_const[4] + source[1];
			active_half[id] = (d_const[4]*(passive_half[id] + passive_half[id+grd])+source[0])/d_coeff_p;


		}
		// If bottom right (SouthEast) corner and black.
		else if (ind_x == (grd-1) && turn == 1)
		{

			cornerSource(d_const[2],d_const[1], source, d_const[4]);
			d_coeff_p = 2.0f * d_const[4] + source[1];
			active_half[id] = (d_const[4]*(passive_half[id] + passive_half[id+grd])+source[0])/d_coeff_p;
		}
		// Bottom row no corner.
		else
		{
			// Check South Boundary Condition.  If it's constant temperature:
			if (d_const[2]>0)
			{
				source[0] = 2.0f * d_const[4] * d_const[2];
				source[1] = 2.0f * d_const[4];
				d_coeff_p = 3.0f * d_const[4] + source[1];
				active_half[id] = (d_const[4]*(passive_half[id]+passive_half[id+grd]+passive_half[id+1])+source[0])/d_coeff_p;

			}
			else
			{
				d_coeff_p = 3.0f * d_const[4];
				active_half[id] = d_const[4] * (passive_half[id]+passive_half[id+grd]+passive_half[id+1])/d_coeff_p;
			}
		}
	}
	// If top row
	else if (ind_y == (int(DIVISIONS)-1))
	{
		// If top right (NorthEast) corner and red.
		if (ind_x == (grd-1) && turn == 0)
		{

			cornerSource(d_const[0],d_const[1], source, d_const[4]);
			d_coeff_p = 2.0f * d_const[4] + source[1];
			active_half[id] = (d_const[4]*(passive_half[id] + passive_half[id-grd])+source[0])/d_coeff_p;

		}
		// If top left (NorthWest) corner and black.
		else if (ind_x == 0 && turn == 1)
		{

			cornerSource(d_const[0],d_const[3], source, d_const[4]);
			d_coeff_p = 2.0f * d_const[4] + source[1];
			active_half[id] = (d_const[4]*(passive_half[id] + passive_half[id-grd])+source[0])/d_coeff_p;


		}
		// Top row no corner.  The top row is the compliment of the bottom row so the operation for seq is reversed.
		else
		{

			// Check North Boundary Condition.  If it's constant temperature:
			if (d_const[0]>0)
			{
				source[0] = 2.0f * d_const[4] * d_const[0];
				source[1] = 2.0f * d_const[4];
				d_coeff_p = 3.0f * d_const[4] + source[1];
				active_half[id] = (d_const[4]*(passive_half[id]+passive_half[id-grd]+passive_half[id+seq])+source[0])/d_coeff_p;
			}
			else
			{
				d_coeff_p = 3.0f * d_const[4];
				active_half[id] = d_const[4]*(passive_half[id]+passive_half[id-grd]+passive_half[id+seq])/d_coeff_p;
			}
		}
	}
	// Check side walls.  This is West when the matrix starts the row, that's when seq is -1.
	else if (ind_x == 0 && seq == -1)
	{
		if (d_const[3]>0)
		{

			source[0] = 2.0f * d_const[4]*d_const[3];
			source[1] = 2.0f * d_const[4];
			d_coeff_p = 3.0f * d_const[4] + source[1];
			active_half[id] = (d_const[4]*(passive_half[id]+ passive_half[id+grd] + passive_half[id-grd])+source[0])/d_coeff_p;


		}
		else
		{

			d_coeff_p = 3.0f * d_const[4];
			active_half[id] = d_const[4]*(passive_half[id]+passive_half[id+grd]+passive_half[id-grd])/d_coeff_p;



		}
	}

	// This is East when the matrix ends the row.
	else if (ind_x == (grd-1) && seq == 1)
	{
		if (d_const[1]>0)
		{

			source[0] = 2.0f * d_const[4]*d_const[1];
			source[1] = 2.0f * d_const[4];
			d_coeff_p = 3.0f * d_const[4] + source[1];
			active_half[id] = (d_const[4]*(passive_half[id] + passive_half[id+grd] + passive_half[id-grd])+source[0])/d_coeff_p;
		}
		else
		{
			d_coeff_p = 3.0f * d_const[4];
			active_half[id] = d_const[4]*(passive_half[id]+passive_half[id+grd]+passive_half[id-grd])/d_coeff_p;
		}
	}
	// Every cell not on an edge or corner.
	else
	{
		d_coeff_p = 4.0f * d_const[4];
		active_half[id] = d_const[4]*(passive_half[id]+passive_half[id+grd]+passive_half[id-grd]+passive_half[id+seq])/d_coeff_p;
	}
	}
	delete[] source;

}



int main()
{

	// Get device properties and set threads to be max thread size.
	// We need the threads to fit the matrix correctly so reject the program if they don't.
	hipDeviceProp_t prop;
	hipGetDeviceProperties( &prop, 0 );
	int mt = prop.maxThreadsPerBlock;
	int thread = int(sqrtf(float(mt)));


	if (int(DIVISIONS)%(2*thread) != 0)
	{
		printf("Error: DIVISIONS must be a multiple of %.i.  That's twice the thread dimension.\n",(2*thread));
		return 0;
	}

	int sz = int(DIVISIONS*DIVISIONS)/2;
	thrust::host_vector<REAL> red(sz);
	thrust::host_vector<REAL> black(sz);
	thrust::host_vector<REAL> temp_c(5);
	REAL ds = (REAL)LENS/((REAL)(DIVISIONS-1));
    REAL A = (REAL)DZ * ds;
	const int y_gr = (int)DIVISIONS/thread;
	const int x_gr = y_gr/2;
	REAL dm2;

    // Get initial conditions
 //   cout << "Provide Boundary conditions for each edge of the slab.\nEnter Constant Temperature in KELVIN\nor a negative number for an insulated boundary:\nNorth: \n";
	//cin >> temp_c[0];
 //   cout << "East: \n";
 //   cin >> temp_c[1];
 //   cout << "South: \n";
 //   cin >> temp_c[2];
 //   cout << "West: \n";
 //   cin >> temp_c[3];

 //   // Get Guess for slab temperature
 //   cout << "Provide a guess Temperature for the slab in Kelvin:\n";
 //   cin >> temp_c[4];

	// For debugging:
	temp_c[0] = 500.;
	temp_c[1] = -9.;
	temp_c[2] = 800.;
	temp_c[3] = -9.;
	temp_c[4] = (REAL)TH_COND * A / ds;
	REAL guess = 650.;

	// Copy the Initial arrays to the GPU.
	thrust::device_vector<REAL> d_red_i(sz,guess);
	thrust::device_vector<REAL> d_red_f(sz,guess);
	thrust::device_vector<REAL> d_black_i(sz,guess);
	thrust::device_vector<REAL> d_black_f(sz,guess);
	thrust::device_vector<REAL> diff_mat(sz);
	thrust::device_vector<REAL> t_2 = temp_c;


	REAL *d_const = thrust::raw_pointer_cast(&t_2[0]);
	REAL *red_cast = thrust::raw_pointer_cast(&d_red_f[0]);
	REAL *black_cast = thrust::raw_pointer_cast(&d_black_f[0]);
//	REAL *red_casti = thrust::raw_pointer_cast(&d_red_i[0]);
//	REAL *black_casti = thrust::raw_pointer_cast(&d_black_i[0]);

	dim3 grids(x_gr,y_gr);
	dim3 threads(thread,thread);
	bool stops = true;
	int iter = 0;
	double wall0 = clock();

	while (stops)
	{

		differencingOperation <<< grids, threads >>> (red_cast, black_cast, d_const, 0);

		hipDeviceSynchronize();

		differencingOperation <<< grids, threads >>> (black_cast, red_cast, d_const, 1);

		hipDeviceSynchronize();

		thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(d_red_i.begin(), d_red_f.begin(), d_black_i.begin(), d_black_f.begin(), diff_mat.begin())),
			thrust::make_zip_iterator(thrust::make_tuple(d_red_i.end(), d_red_f.end(), d_black_i.end(), d_black_f.end(), diff_mat.end())),
			absdiff());


		hipDeviceSynchronize();

		dm2 = thrust::reduce(diff_mat.begin(),diff_mat.end());

		iter++;

		if (((dm2 /REAL(sz*2)) < TOLERANCE) || (iter>1e7))
		{
			stops = false;
		}

		//d_red_i = d_red_f;

		//hipMemcpy(red_casti, red_cast, sz * sizeof(REAL), hipMemcpyDeviceToDevice);
		thrust::copy(d_red_f.begin(), d_red_f.end(), d_red_i.begin());

		//d_black_i = d_black_f;

		//hipMemcpy(black_casti, black_cast, sz * sizeof(REAL), hipMemcpyDeviceToDevice);
		thrust::copy(d_black_f.begin(), d_black_f.end(), d_black_i.begin());

		hipDeviceSynchronize();
		if (iter%100 == 0) cout << "Iteration: " << iter << "dm:" << dm2/REAL(sz*2) << endl;
		//Just to be super obnoxious.
		//ofstream filewrite;
		//filewrite.open("C:\\Users\\Philadelphia\\Documents\\1_SweptTimeResearch\\GaussSeidel\\GaussSeidelCUDA\\GS_outputCUDA.dat", ios::trunc);
		//
  //      for (int n = 0; n < (sz); n++)
  //      {
  //          filewrite << "\n" << d_red_f[n] << "\n" << d_black_i[n];
  //      }
		//filewrite.close();

	}

    double wall1 = clock();
    double timed = (wall1-wall0)/CLOCKS_PER_SEC;

	printf("Outside the loop\n");

	printf("It converged after %d iterations: \n",iter);

	// Write it out!
	/*ofstream filewrite;
	filewrite.open("C:\\Users\\Philadelphia\\Documents\\1_SweptTimeResearch\\GaussSeidel\\GaussSeidelCUDA\\GS_outputCUDA.dat", ios::trunc);
	filewrite << DIVISIONS << "\n" << ds;
    for (int k = 0; k < x_dim; k++)
    {
        for (int n = 0; n < DIVISIONS; n++)
        {
            filewrite << "\n" << red[k][n] << "\n" << black[k][n];
        }
    }*/

    // filewrite.close();

    return 0;
}
