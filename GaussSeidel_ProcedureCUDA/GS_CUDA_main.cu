#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"
#include "hip/hip_runtime_api.h"
#include "hip/driver_types.h"

#include <thrust\reduce.h>
#include <thrust\execution_policy.h>
#include <thrust\device_vector.h>
#include <thrust\host_vector.h>
#include <thrust\for_each.h>
#include <thrust\transform.h>
#include <thrust\iterator\zip_iterator.h>
#include <thrust\copy.h>

#include <stdio.h>
#include <cmath>
#include <ctime>
#include <stdlib.h>
#include <fstream>
#include <iostream>
#include <typeinfo>

using namespace std;

// Now the plate must be square.  #Divisions should be some multiple of 32, preferably some 2^x.
#define LENS       5.
#define TH_COND   16.
#define DZ        .01
#define DIVISIONS 256.
#define TOLERANCE 1.e-5
#define REAL double

struct absdiff
{
    template <typename Tuple>
    __host__ __device__
    void operator()(Tuple t)
    {
        // diffmat = (redi-redf)+blacki-blackf) 
        thrust::get<4>(t) = fabs(thrust::get<0>(t) - thrust::get<1>(t)) + fabs(thrust::get<2>(t) - thrust::get<3>(t));
    }
};

__device__ void cornerSource (REAL BC1, REAL BC2, REAL *source, REAL coff)
{
	printf("Corner Source was called!\n");
    if (BC1>0)
    {
        if (BC2>0)
        {
            source[0] = 2.0f * coff * (BC1 + BC2);
            source[1] = 4.0f * coff;
        }
        else
        {
            source[0] = 2.0f * coff * BC1;
            source[1] = 2.0f * coff;
        }
    }
	
    else if (BC2>0)
    {
        source[0] = 2.0f * coff * BC2;
        source[1] = 2.0f * coff;
    }
    else
    {
        source[0] = 0.0f;
        source[1] = 0.0f;
		
    }
}

__global__ void differencingOperation(REAL *active_half, REAL *passive_half, REAL *d_const, const int turn)
{
	int ind_x = blockIdx.x * blockDim.x + threadIdx.x;
	int ind_y = blockIdx.y * blockDim.y + threadIdx.y;
	int id = ind_x + ind_y * int(DIVISIONS/2);
	int grd;
	grd = int(DIVISIONS)/2;
	int seq;
	int s1 = turn + ind_y;

    REAL d_coeff_p;
    REAL *source = new REAL[2];

	// Negative seq means active half starts first.  Positive seq means passive half starts first.
	if ((s1 & 1) == 0)
	{
		seq = -1;
	}
	else
	{
		seq = 1;
	}

	//printf("Sequence:  id %d seq %d iy: %d s1: %d gridx: %d\n", id, seq, ind_y, s1, grd);
	//printf("Sequence:  id %d North: %.f East: %.f South: %.f West: %.f a: %.8f grid: %d \n",id, d_const[0],d_const[1],d_const[2],d_const[3], d_const[4], grd);
	// If bottom row.
	if (id < DIVISIONS*DIVISIONS*.5)
	{
	if (ind_y == 0)
	{
		// If bottom left (SouthWest) corner and red.
		if (ind_x == 0 && turn == 0)
		{
			cornerSource(d_const[2],d_const[3], source, d_const[4]);
			d_coeff_p = 2.0f * d_const[4] + source[1];	
			active_half[id] = (d_const[4]*(passive_half[id] + passive_half[id+grd])+source[0])/d_coeff_p;
			printf("Southwest activeHalf is now %.4f.\n",active_half[id]);
			printf("Coefficient is %.4f: \n",d_const[4]);
			printf("Active node coefficient is %.4f: \n",d_coeff_p);
			printf("Memory location : %p\n", active_half[id]);
			
		}
		// If bottom right (SouthEast) corner and black.
		else if (ind_x == ((int(DIVISIONS)/2)-1) && turn == 1)
		{
			cornerSource(d_const[2],d_const[1], source, d_const[4]);
			d_coeff_p = 2.0f * d_const[4] + source[1];
			active_half[id] = (d_const[4]*(passive_half[id] + passive_half[id+grd])+source[0])/d_coeff_p;
			printf("Southeast activeHalf is now %.4f.\n",active_half[id]);
		}
		// Bottom row no corner.
		else
		{
			// Check South Boundary Condition.  If it's constant temperature:
			if (d_const[2]>0)
			{
				source[0] = 2.0f * d_const[4] * d_const[2];
				source[1] = 2.0f * d_const[4];
				d_coeff_p = 3.0f * d_const[4] + source[1];
				active_half[id] = (d_const[4]*(passive_half[id]+passive_half[id+grd]+passive_half[id+1])+source[0])/d_coeff_p;

			}
			else
			{
				d_coeff_p = 3.0f * d_const[4];
				active_half[id] = d_const[4] * (passive_half[id]+passive_half[id+grd]+passive_half[id+1])/d_coeff_p;
			}
		}
	}
	// If top row
	else if (ind_y == (int(DIVISIONS)-1))
	{
		// If top right (NorthEast) corner and red.
		if (ind_x == ((int(DIVISIONS)/2)-1) && turn == 0)
		{
			cornerSource(d_const[0],d_const[1], source, d_const[4]);
			d_coeff_p = 2.0f * d_const[4] + source[1];
			active_half[id] = (d_const[4]*(passive_half[id] + passive_half[id-grd])+source[0])/d_coeff_p;	
			printf("Northeast activeHalf is now %.4f.\n",active_half[id]); 
		}
		// If top left (NorthWest) corner and black.
		else if (ind_x == 0 && turn == 1)
		{
			cornerSource(d_const[0],d_const[3], source, d_const[4]);
			d_coeff_p = 2.0f * d_const[4] + source[1];
			active_half[id] = (d_const[4]*(passive_half[id] + passive_half[id-grd])+source[0])/d_coeff_p;
			printf("Northwest activeHalf is now %.4f.\n",active_half[id]); 

		}
		// Top row no corner.  The top row is the compliment of the bottom row so the operation for seq is reversed.
		else
		{
			// Check North Boundary Condition.  If it's constant temperature:
			if (d_const[0]>0)
			{
				source[0] = 2.0f * d_const[4] * d_const[0];
				source[1] = 2.0f * d_const[4];
				d_coeff_p = 3.0f * d_const[4] + source[1];
				active_half[id] = (d_const[4]*(passive_half[id]+passive_half[id-grd]+passive_half[id+seq])+source[0])/d_coeff_p;
			}
			else
			{
				d_coeff_p = 3.0f * d_const[4];
				active_half[id] = d_const[4]*(passive_half[id]+passive_half[id-grd]+passive_half[id+seq])/d_coeff_p;
			}
		}
	}
	// Check side walls.  This is West when the matrix starts the row, that's when seq is -1.
	else if (ind_x == 0 && seq == -1)
	{
		if (d_const[3]>0)
		{
			source[0] = 2.0f * d_const[4]*d_const[3];
			source[1] = 2.0f * d_const[4];
			d_coeff_p = 3.0f * d_const[4] + source[1];	
			//printf("West:  ");		
			//printf("Coefficient is %.4f: \n",d_const[4]);
			//printf("Active node coefficient is %.4f: \n",d_coeff_p);
			
			active_half[id] = (d_const[4]*(passive_half[id]+ passive_half[id+grd] + passive_half[id-grd])+source[0])/d_coeff_p;

			
		}
		else
		{
			d_coeff_p = 3.0f * d_const[4];
			active_half[id] = d_const[4]*(passive_half[id]+passive_half[id+grd]+passive_half[id-grd])/d_coeff_p;
			//printf("The active_half has a value %.5f at [%d][%d].\n",active_half[id],ind_x,ind_y);
		}
	}
	
	// This is East when the matrix ends the row.
	else if (ind_x == ((int(DIVISIONS)/2)-1) && seq == 1)
	{
		if (d_const[1]>0)
		{
			source[0] = 2.0f * d_const[4]*d_const[1];
			source[1] = 2.0f * d_const[4];
			d_coeff_p = 3.0f * d_const[4] + source[1];
			active_half[id] = (d_const[4]*(passive_half[id]+ passive_half[id+grd] + passive_half[id-grd])+source[0])/d_coeff_p;
		}
		else
		{
			d_coeff_p = 3.0f * d_const[4];
			active_half[id] = d_const[4]*(passive_half[id]+passive_half[id+grd]+passive_half[id-grd])/d_coeff_p;
		}
	}
	// Every cell not on an edge or corner.
	else
	{
		d_coeff_p = 4.0f * d_const[4];
		active_half[id] = d_const[4]*(passive_half[id]+passive_half[id+grd]+passive_half[id-grd]+passive_half[id+seq])/d_coeff_p;
	}
	}
}

int main()
{

	// Test copy vector.
	hipDeviceSynchronize();
	//Test even odd

	// Get device properties and set threads to be max thread size.  
	// We need the threads to fit the matrix correctly so reject the program if they don't.
	hipDeviceProp_t prop;
	hipGetDeviceProperties( &prop, 0 );
	int mt = prop.maxThreadsPerBlock;
	int thread = int(sqrtf(float(mt)));
	cout << "Number of threads" << thread << "\n";

	if (int(DIVISIONS)%(2*thread) != 0)
	{
		printf("Error: DIVISIONS must be a multiple of %.i.  That's twice the thread dimension.\n",(2*thread));
		return 0;
	}

	int sz = int(DIVISIONS*DIVISIONS)/2;
	thrust::host_vector<REAL> red(sz);
	thrust::host_vector<REAL> black(sz);
	thrust::host_vector<REAL> temp_c(5);
	REAL ds = (REAL)LENS/((REAL)(DIVISIONS-1));
    REAL A = (REAL)DZ * ds;
	const int y_gr = (int)DIVISIONS/thread;
	const int x_gr = y_gr/2;
	REAL dm2; 
	cout << "The x grid dimension: " << x_gr << " The y grid dimension: " << y_gr << endl;
	
    // Get initial conditions
 //   cout << "Provide Boundary conditions for each edge of the slab.\nEnter Constant Temperature in KELVIN\nor a negative number for an insulated boundary:\nNorth: \n";
	//cin >> temp_c[0];
 //   cout << "East: \n";
 //   cin >> temp_c[1];
 //   cout << "South: \n";
 //   cin >> temp_c[2];
 //   cout << "West: \n";
 //   cin >> temp_c[3];

 //   // Get Guess for slab temperature
 //   cout << "Provide a guess Temperature for the slab in Kelvin:\n";
 //   cin >> temp_c[4];

	// For debugging:
	temp_c[0] = 500.;
	temp_c[1] = 740.;
	temp_c[2] = 900.;
	temp_c[3] = -9.;
	temp_c[4] = (REAL)TH_COND * A / ds;
	REAL guess = 600.;
	
	// Copy the Initial arrays to the GPU.
	thrust::device_vector<REAL> d_red_i(sz,guess);
	thrust::device_vector<REAL> d_red_f(sz,guess);
	thrust::device_vector<REAL> d_black_i(sz,guess);
	thrust::device_vector<REAL> d_black_f(sz,guess);
	thrust::device_vector<REAL> diff_mat(sz);
	thrust::device_vector<REAL> t_2 = temp_c;



	dim3 grids(x_gr,y_gr);
	dim3 threads(thread,thread);

	bool stops = true;
	int iter = 0;

	while (stops)
	{
		REAL *d_const = thrust::raw_pointer_cast(&t_2[0]);
		REAL *red_cast = thrust::raw_pointer_cast(&d_red_f[0]);
		REAL *black_cast = thrust::raw_pointer_cast(&d_black_f[0]);
		REAL *red_casti = thrust::raw_pointer_cast(&d_red_i[0]);
		REAL *black_casti = thrust::raw_pointer_cast(&d_black_i[0]);

		differencingOperation <<< grids, threads >>> (red_cast, black_cast, d_const, 0);

		printf("\nNumber One!\n");

		differencingOperation <<< grids, threads >>> (black_cast, red_cast, d_const, 1);

		printf("\nNumber Two!\n");

		thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(d_red_i.begin(), d_red_f.begin(), d_black_i.begin(), d_black_f.begin(), diff_mat.begin())),
			thrust::make_zip_iterator(thrust::make_tuple(d_red_i.end(), d_red_f.end(), d_black_i.end(), d_black_f.end(), diff_mat.end())),
			absdiff());

		printf("\nAnd the subtraction!\n");

		dm2 = thrust::reduce(diff_mat.begin(),diff_mat.end());

		iter++;
		printf("\nAnd the reduction!\n");

		if ((dm2 /sz < TOLERANCE) || (iter>1e7))
		{
			stops = false;
		}	

		cout << "Fifth brick red final: " << d_red_f[5] << " Fifth brick black final: " << d_black_f[5] << endl;
		cout << "And the check!\n";
		cout << "dm2 is " << dm2 << endl;
		cout << "Diff Matrix First element is:  " << diff_mat[0] << endl;

		//Hmm.  It doesn't look like they've changed.
		//Yep.  Here's your problem.

		// There should be three ways to do this.
		//d_red_i = d_red_f;
		//thrust::copy(d_red_f.begin(), d_red_f.end(), d_red_i.begin());

		hipMemcpy(red_casti, red_cast, d_red_f.size() * sizeof(float), hipMemcpyDeviceToDevice); //This one just doesn't do anything
		
		cout << "The first copy finished!" << endl;
		cout << "Initial red: " << endl;

		//d_black_i = d_black_f;
		//thrust::copy(d_black_f.begin(), d_black_f.end(), d_black_i.begin());
		hipMemcpy(black_casti, black_cast, d_red_f.size() * sizeof(float), hipMemcpyDeviceToDevice);
		cout << "The second copy finished!" << endl;
		cout << d_black_i[0] << endl;
		
	}

	printf("Outside the loop\n");

	printf("It converged after %.f iterations: \n",iter);
	
	
	// Write it out!
	/*ofstream filewrite;
	filewrite.open("C:\\Users\\Philadelphia\\Documents\\1_SweptTimeResearch\\GaussSeidel\\GaussSeidelCUDA\\GS_outputCUDA.dat", ios::trunc);
	filewrite << DIVISIONS << "\n" << ds;

    for (int k = 0; k < x_dim; k++)
    {
        for (int n = 0; n < DIVISIONS; n++)
        {
            filewrite << "\n" << red[k][n] << "\n" << black[k][n];
        }
    }*/

    // filewrite.close();

    return 0;
}